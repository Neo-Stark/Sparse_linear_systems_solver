//
// Created by Francisco José González García.
// Copyright (c) 2020 Universidad de Granada. All rights reserved.
//

#include <iostream>
#include "jacobiCuda.h"
#include "kernels.cuh"
#include "utilidades.h"

double *jacobi_CUDA::multiplicacionMV() {
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size{};
    const unsigned int warp_size = 32; /// One warp per row
    grid_size.x = (warp_size * getFilas() + block_size.x - 1) / block_size.x;

    hipMemcpy(x_d, x.data(), getColumnas() * sizeof(double), hipMemcpyHostToDevice);

    matrix_vector_multiplication<double><<<grid_size, block_size>>>(A, col_ind, row_ptr, x_d, y_d, getFilas());

    hipMemcpy(y, y_d, getFilas() * sizeof(double), hipMemcpyDeviceToHost);

    return y;
}

double jacobi_CUDA::norma() {
    double r_max = utilidades::reduce_max_CUDA(r_d, getFilas(), BLOCK_SIZE);
    double x_max = utilidades::reduce_max_CUDA(x_d, getFilas(), BLOCK_SIZE);
    double norma = r_max / x_max;
//    cout << "r_max: " << r_max;
//    cout << "  x_max: " << x_max << endl;
//    cout << " norma: " << norma << endl;
    return norma;
}

void jacobi_CUDA::obtenerNuevaX() {
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size{};
    grid_size.x = (getFilas() + block_size.x - 1) / block_size.x;
    hipMemcpy(r_d, r, getFilas() * sizeof(double), hipMemcpyHostToDevice);
    nuevaX<double><<<grid_size, block_size>>>(x.size(), x_d, r_d);
}

void jacobi_CUDA::actualizaX() {
    hipMemcpy(x.data(), x_d, getColumnas() * sizeof(double), hipMemcpyDeviceToHost);
}

jacobi_CUDA::jacobi_CUDA(const CSR &m, const vector<double> &aprox_inicial, const int &block_size_arg)
        : jacobi(m, aprox_inicial),
          BLOCK_SIZE(block_size_arg) {
    hipMalloc(&r_d, sizeof(double) * getFilas());
    hipMalloc(&A, sizeof(double) * matriz.getVal().size());
    hipMalloc(&col_ind, sizeof(int) * matriz.getColInd().size());
    hipMalloc(&row_ptr, sizeof(int) * matriz.getRowPtr().size());
    hipMalloc(&x_d, sizeof(double) * getFilas());
    hipMalloc(&y_d, sizeof(double) * getFilas());
    hipMalloc(&inversa_d, sizeof(double) * getFilas());

    hipMemcpy(A, matriz.getVal().data(), matriz.getVal().size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(col_ind, matriz.getColInd().data(), matriz.getColInd().size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_ptr, matriz.getRowPtr().data(), matriz.getRowPtr().size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(inversa_d, inversa, getFilas() * sizeof(double), hipMemcpyHostToDevice);
}

jacobi_CUDA::jacobi_CUDA(const CSR &m, const int &block_size_arg) :
        jacobi_CUDA(m, vector<double>(m.getFilas(), 1), block_size_arg) {}

jacobi_CUDA::~jacobi_CUDA() {
    hipFree(A);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(r_d);
    hipFree(col_ind);
    hipFree(row_ptr);
    hipFree(inversa_d);
};
