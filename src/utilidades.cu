#include "hip/hip_runtime.h"
#pragma clang diagnostic push
#pragma ide diagnostic ignored "openmp-use-default-none"
//
// Created by Francisco José González García.
// Copyright (c) 2020 Universidad de Granada. All rights reserved.
//

#include "utilidades.h"
#include "algorithm"
#include "kernels.cuh"
#include <sys/time.h>


using namespace std;

double utilidades::reduce_max_OMP(const double *v, int n) {
    double maximo = -1e36;
#pragma omp parallel for reduction (max : maximo)
    for (int i = 0; i < n; i++) {
        maximo = max(maximo, abs(v[i]));
    }
    return maximo;
}

double utilidades::reduce_max_CUDA(const double *d_vi, int n, const int BLOCK_SIZE) {
    dim3 block(BLOCK_SIZE);
    dim3 grid = (n / 2 + block.x) / block.x;
    auto smemSize = block.x * sizeof(double);
    double *d_vo, *h_vo = new double[grid.x];
    hipMalloc(&d_vo, sizeof(double) * grid.x);
    switch (BLOCK_SIZE) {
        case 1024:
            reduction_max<double, 1024><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 512:
            reduction_max<double, 512><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 256:
            reduction_max<double, 256><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 128:
            reduction_max<double, 128><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 64:
            reduction_max<double, 64><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 32:
            reduction_max<double, 32><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
    }
    hipMemcpy(h_vo, d_vo, sizeof(double) * grid.x, hipMemcpyDeviceToHost);
    double maximo = 0.0;
    for (int i = 0; i < grid.x; i++) maximo = max(maximo, h_vo[i]);

    hipFree(d_vo);
    free(h_vo);
    return maximo;
}

double utilidades::reduce_max_sec(const double *v, int n) {
    double maximo = 0;
    for (int line = 0; line < n; ++line) {
        maximo = (fabs(v[line]) > maximo ? fabs(v[line]) : maximo);
    }
    return maximo;
}

double utilidades::cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1e-6);
}


#pragma clang diagnostic pop