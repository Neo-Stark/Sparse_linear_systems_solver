#include "hip/hip_runtime.h"
//
// Created by Francisco José González García.
// Copyright (c) 2020 Universidad de Granada. All rights reserved.
//

#include "jacobi.h"
#include <kernels.cuh>
#include <iostream>
#include <omp.h>
#include "utilidades.h"


jacobi::jacobi(const CSR &m, const vector<double> &aprox_inicial, const int &block_size_arg)
        : x(aprox_inicial), BLOCK_SIZE(block_size_arg), matriz(m) {
    if (m.getFilas() == 0)
        throw std::invalid_argument("la matriz no está inicializada");

    y = new double[getFilas()];
    r = new double[getFilas()];
    diagonal = calculaDiagonal();
    inversa = inversaDiagonal();
    hipMalloc(&r_d, sizeof(double) * getFilas());
    hipMalloc(&A, sizeof(double) * matriz.getVal().size());
    hipMalloc(&col_ind, sizeof(int) * matriz.getColInd().size());
    hipMalloc(&row_ptr, sizeof(int) * matriz.getRowPtr().size());
    hipMalloc(&x_d, sizeof(double) * getFilas());
    hipMalloc(&y_d, sizeof(double) * getFilas());
    hipMalloc(&inversa_d, sizeof(double) * getFilas());

    hipMemcpy(A, matriz.getVal().data(), matriz.getVal().size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(col_ind, matriz.getColInd().data(), matriz.getColInd().size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_ptr, matriz.getRowPtr().data(), matriz.getRowPtr().size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(inversa_d, inversa, getFilas() * sizeof(double), hipMemcpyHostToDevice);
}

jacobi::jacobi(const CSR &m, const int &block_size_arg) :
        jacobi(m, vector<double>(m.getFilas(), 1), block_size_arg) {}


double jacobi::norma() {
    double sum_cuadrados = 0;
    for (size_t i = 0; i < getFilas(); i++) {
        sum_cuadrados += pow(r[i], 2);
    }
    auto raiz = sqrt(sum_cuadrados);
    return raiz;
}

double *jacobi::calculaDiagonal() {
    auto *D = new double[matriz.getFilas()];
    for (int i = 0; i < getFilas(); i++) {
        D[i] = 0;
        for (int j = matriz.getRowPtr()[i]; j < matriz.getRowPtr()[i + 1]; ++j) {
            if (matriz.getColInd()[j] == i) D[i] = matriz.getVal()[j];
        }
    }
    return D;
}

int jacobi::getFilas() {
    return matriz.getFilas();
}

double *jacobi::getDiagonal() const {
    return diagonal;
}

double *jacobi::inversaDiagonal() {
    auto inver_diag = new double[getFilas()];
    for (int i = 0; i < getFilas(); i++) {
        if (diagonal[i] != 0)
            inver_diag[i] = 1.0 / diagonal[i];
        else
            inver_diag[i] = 0;
    }
    return inver_diag;
}

double *jacobi::getInversa() const {
    return inversa;
}

void jacobi::calculaResiduo(const double *b) {
    for (int i = 0; i < getFilas(); i++) {
        r[i] = (b[i] - y[i]) * getInversa(i);
    }
}

int jacobi::getColumnas() {
    return matriz.getColumnas();
}

double *jacobi::multiplicacionMV_CUDA() {
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size{};
    const unsigned int warp_size = 32; /// One warp per row
    grid_size.x = (warp_size * getFilas() + block_size.x - 1) / block_size.x;

    hipMemcpy(x_d, x.data(), getColumnas() * sizeof(double), hipMemcpyHostToDevice);

    matrix_vector_multiplication<double><<<grid_size, block_size>>>(A, col_ind, row_ptr, x_d, y_d, getFilas());

    hipMemcpy(y, y_d, getFilas() * sizeof(double), hipMemcpyDeviceToHost);

    return y;
}

double *jacobi::getY() {
    return y;
}

double &jacobi::getX(int i) {
    return x[i];
}

const vector<double> &jacobi::getX() const {
    return x;
}

double jacobi::getY(int i) {
    return y[i];
}

double jacobi::getInversa(int i) {
    return inversa[i];
}

double *jacobi::multiplicacionMV_OMP() {
#pragma omp parallel
    {
//        cout << "multiplicando - Hebra " << omp_get_thread_num() << endl;
#pragma omp for
        for (int i = 0; i < getFilas(); i++) {
            const unsigned int row_start = matriz.getRowPtr()[i];
            const unsigned int row_end = matriz.getRowPtr()[i + 1];
            y[i] = 0;
            for (auto j = row_start; j < row_end; j++) {
                y[i] += matriz.getVal()[j] * x[matriz.getColInd()[j]];
            }
        }
    }
    return y;
}

void jacobi::obtenerNuevaX() {
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size{};
    grid_size.x = (getFilas() + block_size.x - 1) / block_size.x;
    hipMemcpy(r_d, r, getFilas() * sizeof(double), hipMemcpyHostToDevice);
    nuevaX<double><<<grid_size, block_size>>>(x.size(), x_d, r_d);
}

void jacobi::actualizaX() {
    hipMemcpy(x.data(), x_d, getColumnas() * sizeof(double), hipMemcpyDeviceToHost);
}

double jacobi::norma_CUDA() {
    double r_max = utilidades::reduce_max_CUDA(r_d, getFilas(), BLOCK_SIZE);
    double x_max = utilidades::reduce_max_CUDA(x_d, getFilas(), BLOCK_SIZE);
    double norma = r_max / x_max;
//    cout << "r_max: " << r_max;
//    cout << "  x_max: " << x_max << endl;
//    cout << " norma: " << norma << endl;
    return norma;
}

double jacobi::norma_OMP() {
    double r_max = utilidades::reduce_max_OMP(r, getFilas());
    double x_max = utilidades::reduce_max_OMP(x.data(), getFilas());
    double norma = r_max / x_max;
    return norma;
}

jacobi::~jacobi() {
    free(y);
    free(r);
    free(diagonal);
    free(inversa);
    hipFree(A);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(r_d);
    hipFree(col_ind);
    hipFree(row_ptr);
    hipFree(inversa_d);
}

double *jacobi::getR() const {
    return r;
}

double jacobi::getR(int i) {
    return r[i];
}



