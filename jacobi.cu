#include "hip/hip_runtime.h"
//
// Created by Francisco José González García.
// Copyright (c) 2020 Universidad de Granada. All rights reserved.
//

#include "jacobi.h"
#include <kernels.cuh>
#include <iostream>
#include <omp.h>


jacobi::jacobi(const CSR &m, const vector<double> &aprox_inicial, const int &block_size_arg)
        : x(aprox_inicial), BLOCK_SIZE(block_size_arg), matriz(m) {
    if (m.getFilas() == 0)
        throw std::invalid_argument("la matriz no está inicializada");

    y = new double[getFilas()];
    r = new double[getFilas()];
    diagonal = calculaDiagonal();
    inversa = inversaDiagonal();
    hipMalloc(&r_d, sizeof(double) * getFilas());
    hipMalloc(&A, sizeof(double) * matriz.getVal().size());
    hipMalloc(&col_ind, sizeof(int) * matriz.getColInd().size());
    hipMalloc(&row_ptr, sizeof(int) * matriz.getRowPtr().size());
    hipMalloc(&x_d, sizeof(double) * getFilas());
    hipMalloc(&y_d, sizeof(double) * getFilas());
    hipMalloc(&inversa_diag, sizeof(double) * getFilas());

    hipMemcpy(A, matriz.getVal().data(), matriz.getVal().size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(col_ind, matriz.getColInd().data(), matriz.getColInd().size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(row_ptr, matriz.getRowPtr().data(), matriz.getRowPtr().size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(inversa_diag, inversa, getFilas() * sizeof(double), hipMemcpyHostToDevice);
}

jacobi::jacobi(const CSR &m, const int &block_size_arg) :
        jacobi(m, vector<double>(m.getFilas(), 1), block_size_arg) {}


double jacobi::norma() {
    double sum_cuadrados = 0;
    for (size_t i = 0; i < getFilas(); i++) {
        sum_cuadrados += pow(r[i], 2);
    }
    auto raiz = sqrt(sum_cuadrados);
    return raiz;
}

double *jacobi::calculaDiagonal() {
    auto *D = new double[matriz.getFilas()];
    for (int i = 0; i < getFilas(); i++) {
        D[i] = 0;
        for (int j = matriz.getRowPtr()[i]; j < matriz.getRowPtr()[i + 1]; ++j) {
            if (matriz.getColInd()[j] == i) D[i] = matriz.getVal()[j];
        }
    }
    return D;
}

int jacobi::getFilas() {
    return matriz.getFilas();
}

double *jacobi::getDiagonal() const {
    return diagonal;
}

double *jacobi::inversaDiagonal() {
    auto inver_diag = new double[getFilas()];
    for (int i = 0; i < getFilas(); i++) {
        if (diagonal[i] != 0)
            inver_diag[i] = 1.0 / diagonal[i];
        else
            inver_diag[i] = 0;
    }
    return inver_diag;
}

double *jacobi::getInversa() const {
    return inversa;
}

void jacobi::calculaResiduo(const double *b) {
    for (int i = 0; i < getFilas(); i++) {
        r[i] = b[i] - y[i];
    }
}

int jacobi::getColumnas() {
    return matriz.getColumnas();
}

double *jacobi::multiplicacionMV_CUDA() {
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size{};
    const unsigned int warp_size = 32; /// One warp per row
    grid_size.x = (warp_size * getFilas() + block_size.x - 1) / block_size.x;

    hipMemcpy(x_d, x.data(), getColumnas() * sizeof(double), hipMemcpyHostToDevice);

    csr_spmv_vector_kernel<double><<<grid_size, block_size>>>(getFilas(), col_ind, row_ptr, A, x_d, y_d);

    hipMemcpy(y, y_d, getFilas() * sizeof(double), hipMemcpyDeviceToHost);

    return y;
}

double *jacobi::getY() {
    return y;
}

double &jacobi::getX(int i) {
    return x[i];
}

const vector<double> &jacobi::getX() const {
    return x;
}

double jacobi::getY(int i) {
    return y[i];
}

double jacobi::getInversa(int i) {
    return inversa[i];
}

double *jacobi::multiplicacionMV_OMP() {
#pragma omp parallel num_threads(4)
    cout << "multiplicando - Hebra " << omp_get_thread_num() << endl;
#pragma omp for
    for (int i = 0; i < getFilas(); i++) {
        const unsigned int row_start = matriz.getRowPtr()[i];
        const unsigned int row_end = matriz.getRowPtr()[i + 1];
        y[i] = 0;
        for (auto j = row_start; j < row_end; j++) {
            y[i] += matriz.getVal()[j] * x[matriz.getColInd()[j]];
        }
    }
    return y;
}

void jacobi::obtenerNuevaX() {
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size{};
    grid_size.x = (getFilas() + block_size.x - 1) / block_size.x;
//    cout << "r: ";
//    for(int i = 0; i < getFilas(); i++) cout << r[i] << " ";
//    cout << endl;
    hipMemcpy(r_d, r, getFilas() * sizeof(double), hipMemcpyHostToDevice);

    kernelNuevaX<double><<<grid_size, block_size>>>(x.size(), x_d, r_d, inversa_diag);
}

void jacobi::actualizaX() {
    hipMemcpy(x.data(), x_d, getColumnas() * sizeof(double), hipMemcpyDeviceToHost);
}

double jacobi::normaInfinito_r() {
    double r_max = reduce_max_CUDA(r_d, getFilas());
    cout << "r_max: " << r_max << endl;
    double x_max = reduce_max_CUDA(x_d, getFilas());
    cout << "x_max: " << x_max << endl;
    double norma = r_max / x_max;
    cout << " norma: " << norma << endl;
    return norma;
}

jacobi::~jacobi() {
    free(y);
    free(r);
    free(diagonal);
    free(inversa);
    hipFree(A);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(r_d);
    hipFree(col_ind);
    hipFree(row_ptr);
    hipFree(inversa_diag);
}

double jacobi::reduce_max_CUDA(const double *d_vi, const int n) const {
    dim3 block(BLOCK_SIZE);
    dim3 grid = (n / 2 + block.x - 1) / block.x;
    auto smemSize = block.x * sizeof(double);
    double *d_vo, *h_vo = new double[grid.x];
    hipMalloc(&d_vo, sizeof(double) * grid.x);
    switch (BLOCK_SIZE) {
        case 1024:
            reduce_max<double, 1024><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 512:
            reduce_max<double, 512><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 256:
            reduce_max<double, 256><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 128:
            reduce_max<double, 128><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 64:
            reduce_max<double, 64><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
        case 32:
            reduce_max<double, 32><<< grid, block, smemSize >>>(d_vi, d_vo, n);
            break;
    }
    hipMemcpy(h_vo, d_vo, sizeof(double) * grid.x, hipMemcpyDeviceToHost);
    double maximo = 0.0;
    for (int i = 0; i < grid.x; i++) maximo = max(maximo, h_vo[i]);

    hipFree(d_vo);
    free(h_vo);
    return maximo;
}

double *jacobi::getR() const {
    return r;
}

double jacobi::getR(int i) {
    return r[i];
}



